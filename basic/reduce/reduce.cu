#include "hip/hip_runtime.h"

#include "hiprand/hiprand.h"
#include <iostream>
using namespace std;

__global__ void sumSingleBlock(int *d)
{
    int tid = threadIdx.x;
    for (int tc = blockDim.x, stepSize = 1; tc > 0; tc >>=1, stepSize <<= 1)
    {
        if (tid < tc)
        {
            int pa = tid * stepSize * 2;
            int pb = pa + stepSize;
            d[pa] += d[pb];
        }
    }
}

__global__ void sumSingleBlock_shm(int *d)
{
    extern __shared__ int dcopy[];
    int tid = threadIdx.x;

    dcopy[tid*2] = d[tid*2];
    dcopy[tid*2+1] = d[tid*2+1];
    for (int tc = blockDim.x, stepSize = 1; tc > 0; tc >>=1, stepSize <<= 1)
    {
        if (tid < tc)
        {
            int pa = tid * stepSize * 2;
            int pb = pa + stepSize;
            dcopy[pa] += dcopy[pb];
        }
    }
    if (tid == 0) 
    {
        d[0] = dcopy[0];
    }
}

int main()
{
    const int count = 32;
    const size_t size = count * sizeof(int);
    int h[count];
    for (int i=0; i<count; ++i)
    {
        h[i] = i+1;
    }

    int *d;
    hipMalloc(&d, size);
    hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    sumSingleBlock<<<1, count/2>>>(d);

    int result;
    hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);
    //cudaFree(d);
    std::cout << "use global mem:" << result << std::endl;

    hipMemcpy(d, h, size, hipMemcpyHostToDevice);
    sumSingleBlock_shm<<<1, count/2, count>>>(d);

    hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "use shared mem:" << result << std::endl;

    hipFree(d);
}