#include "hip/hip_runtime.h"
#include <stdio.h>
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "utils.h"

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //TODO
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion

  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, then use that to
  //calculate a 1D offset
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  //if (blockIdx.y == 0 && blockIdx.x == 30) printf("got\n");
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= numCols || y >= numRows){
      return;
  }
  int pos = y * numCols + x;
  uchar4 rgba = rgbaImage[y * numCols + x];
  float channelSum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
  greyImage[pos]= channelSum;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  const int block_w = 32;
  //printf ("r:%d c:%d rb:%d cb:%d\n", numRows, numCols, numRows/block_w, numCols/block_w);
  std::cout << "r:" << numRows 
       << "c:" << numCols 
       << "rb:" << numRows/block_w 
       << "cb:" << numCols/block_w;
  const dim3 blockSize(block_w, block_w, 1);  //TODO
  const dim3 gridSize( numCols/block_w + 1, numRows/block_w + 1, 1);  //TODO
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}
