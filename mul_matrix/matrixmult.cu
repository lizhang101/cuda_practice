#include <cstdlib>
#include <stdio.h>
#include <cassert>
#include <hip/hip_runtime.h>
/*
   Naive implementation.
   Allocate one thread for one element in result matrix, processing dot(Arow, Bcol);
*/
__global__ void kMatrixMul0 (float *d_res, 
                               float *d_mat1, int m1, int m2,
                               float *d_mat2, int n1, int n2){
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int y = blockDim.y*blockIdx.y + threadIdx.y;
    if (x >= n2 || y >= m1){return;}
    float sum=0.0;
    for (int i=0; i<m2; i++){
//    	if (blockIdx.x == 0 && threadIdx.x == 0 && blockIdx.y == 0 && threadIdx.y == 0) {
//    		printf("sum:%f mat1:%f mat2:%f i:%d\n", sum, d_mat1[y*m2+i], d_mat2[i*n2+x], i);
//    	}
        sum += d_mat1[y*m2+i] * d_mat2[i*n2+x];
    }
    d_res[n2*y+x] = sum;
};

/* Using shared memory 
*/
template<int BLOCK_SIZE>
__global__ void kMatrixMul1 (float *d_res, 
                               float *d_mat1, int m1, int m2,
                               float *d_mat2, int n1, int n2) {
    //assume squre block
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    //trivial opt: calculate these in CPU, shared by all blocks
    int aStart = blockDim.y*blockIdx.y * m2;
    int bStart = blockDim.x*blockIdx.x;
    int bStep = blockDim.y*n2;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float c = 0.0f;
    for (int a = aStart, b = bStart; a < m2; a += blockDim.x, b += bStep) {
        //load data
        As[ty][tx] = d_mat1[a + m2 * ty + tx];   
        Bs[ty][tx] = d_mat2[b + n2 * ty + tx];
        __syncthreads();
        for (int k = 0; k<blockDim.x; k++){
            c += As[ty][k] * Bs[k][tx];
        }
    }
    d_res[(blockDim.y*blockIdx.y+ty)*n2+blockDim.x*blockIdx.x+tx] = c;
    //__syncthreads();
}


float* MatrixMultGPU0(float *mat1, int m1, int m2, float *mat2, int n1, int n2){
    float *d_res, *d_mat1, *d_mat2;
    hipEvent_t start, end;
    hipError_t error;
    error = hipEventCreate(&start);
    error = hipEventCreate(&end);



    //malloc the device memory for matrices 
    hipError_t result = hipMalloc((void**)&d_res, sizeof(float)*m1*n2);
    result = hipMalloc((void**)&d_mat1, sizeof(float)*m1*m2);
    assert (result == hipSuccess);
    result = hipMalloc((void**)&d_mat2, sizeof(float)*n1*n2);
    assert (result == hipSuccess);

    //init source matrices in device memory
    result = hipMemcpy(d_mat1, mat1, sizeof(float)*m1*m2, hipMemcpyHostToDevice);
    assert (result == hipSuccess);
    result = hipMemcpy(d_mat2, mat2, sizeof(float)*n1*n2, hipMemcpyHostToDevice);
    assert (result == hipSuccess);

    hipEventRecord(start, NULL);

    int N = 32;

    dim3 block_size(N, N);
    //grid width in blocks
    int grid_wib = ceil(float(n2)/float(N));
    //grid height in blocks
    int grid_hib = ceil(float(m1)/float(N));
    dim3 grid_size(grid_wib, grid_hib);

    //naive version
    //kMatrixMul0<<<grid_size, block_size>>>(d_res, d_mat1, m1, m2, d_mat2, n1, n2);

    //sharedMem version
    kMatrixMul1<32><<<grid_size, block_size>>>(d_res, d_mat1, m1, m2, d_mat2, n1, n2);

    //copy back the multiplication result
    float* res = new float[m1*n2];
    result = hipMemcpy(res, d_res, sizeof(float)*m1*n2, hipMemcpyDeviceToHost);
    assert (result == hipSuccess);

    hipEventRecord(end, NULL);
    error = hipEventSynchronize(end);

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, end);
    printf("calculation Time:%f ms\n", msecTotal);

    hipFree(d_res);
    hipFree(d_mat1);
    hipFree(d_mat2);
    return res;
}

float* MatrixMulCPU(float *mat1, int m1, int m2, float *mat2, int n1, int n2){
    assert(m2 == n1 && "matrx a's cols != matrix b's rows");
    
    float* res = new float[m1*n2] ;
    for (int i = 0; i < m1; i++) {
        for (int j = 0; j < n2; j++) {
            res[i*m1 + j] = 0;
            for (int x = 0; x < m2; x++) {
                res[i*m1+j] += mat1[i*m1+x] * mat2[x*n1+j];
            }
        }
    }
    return res;

}

float CompareMatrix(float mat1[], float mat2[], int m1, int m2){
    float err = 0;
    for (int x=0; x<m1; x++){
        for (int y=0; y<m2; y++){
            err += mat1[m2*y+x] - mat2[m2*y+x];
        }
    }
    return err;
    /*
    if (err > 0.1){
        printf("matrix comparison failed.error:%f\n", err); 
        return false;
    }
    return true;
    */
}

void FillMatrix(float mat[], int m1, int m2, float d=1.0){
    for (int i=0; i<m1; i++){
        for (int j=0; j<m2; j++){
            if (d<0.0) {
                mat[i*m1+j] = static_cast<float>(rand())/static_cast<float>(RAND_MAX) * 10.0;
            } else {
                mat[i*m1+j] = d;
            }
        }
    }
}
void PrintMatrix(float mat[], int m1, int m2){
    for (int i=0; i<m1; i++){
        for (int j=0; j<m2; j++){
            printf("%f,", mat[i*m1+j]);
        }
        printf("\n");
    }
}


int main(int argc, char *argv[]){
    int m1 = 1024;
    int m2;
    int n1 = m2 = 1024;
    int n2 = 1024;
    float *matrix_a = new float[m1*m2];
    float *matrix_b = new float[n1*n2];
    FillMatrix((float*)matrix_a, m1, m2);
    FillMatrix((float*)matrix_b, n1, n2);
    //float *ref = MatrixMulCPU((float*)matrix_a, m1, m2, (float*)matrix_b, n1, n2);
    float *res = MatrixMultGPU0((float*)matrix_a, m1, m2, (float*)matrix_b, n1, n2);
    //printf("error:%f\n", CompareMatrix(ref, res, m1, n2));
    //PrintMatrix(res, m1, m2);
    delete[] matrix_a;
    delete[] matrix_b;
    delete res;
    //delete ref;
}
