#include <cstdlib>
#include <stdio.h>
#include <cassert>
#include <hip/hip_runtime.h>

__global__ void kMatrixMul0 (float *d_res, 
                               float *d_mat1, int m1, int m2,
                               float *d_mat2, int n1, int n2){
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int y = blockDim.y*blockIdx.y + threadIdx.y;
    int m = gridDim.x;
    if (x >= n2 || y >= m1){return;}
    float sum;
    for (int i=0; i<m; i++){
        sum += d_mat1[y*m2+i] * d_mat2[i*n2+x];
    }
    d_res[n2*y+x] = sum;
};

float* MatrixMultGPU0(float *mat1, int m1, int m2, float *mat2, int n1, int n2){
    float *d_res, *d_mat1, *d_mat2;
    //malloc the device memory for matrices 
    hipError_t result = hipMalloc((void**)&d_res, sizeof(float)*m1*n2);
    result = hipMalloc((void**)&d_mat1, sizeof(float)*m1*m2);
    result = hipMalloc((void**)&d_mat2, sizeof(float)*n1*n2);

    //init source matrices in device memory
    result = hipMemcpy(d_mat1, mat1, sizeof(float)*m1*m2, hipMemcpyHostToDevice);
    result = hipMemcpy(d_mat2, mat2, sizeof(float)*n1*n2, hipMemcpyHostToDevice);

    int N = 16;

    dim3 block_size(N, N);
    //grid width in blocks
    int grid_wib = ceil(float(m1)/float(N));
    //grid height in blocks
    int grid_hib = ceil(float(n2)/float(N));
    dim3 grid_size(grid_wib, grid_hib);
    kMatrixMul0<<<grid_size, block_size>>>(d_res, d_mat1, m1, m2, d_mat2, n1, n2);
    //copy back the multiplication result
    float* res = new float[m1*n2];
    result = hipMemcpy(res, d_res, sizeof(float)*m1*n2, hipMemcpyDeviceToHost);
    hipFree(d_res);
    hipFree(d_mat1);
    hipFree(d_mat2);
    return res;
}

float* MatrixMulCPU(float *mat1, int m1, int m2, float *mat2, int n1, int n2){
    assert(m2 == n1 && "matrx a's cols != matrix b's rows");
    
    float* res = new float[m1*n2] ;
    for (int i = 0; i < m1; i++) {
        for (int j = 0; j < n2; j++) {
            res[i*m1 + j] = 0;
            for (int x = 0; x < m2; x++) {
                res[i*m1+j] += mat1[i*m1+x] * mat2[x*n1+j];
            }
        }
    }
    return res;

}

bool CompareMatrix(float mat1[], float mat2[], int m1, int m2){
    float err = 0;
    for (int x=0; x<m1; x++){
        for (int y=0; y<m2; y++){
            err += mat1[m2*y+x] - mat2[m2*y+x];
        }
    }
    if (err > 0.1){
        printf("matrix comparison failed.error:%f\n", err); 
        return false;
    }
    return true;
}

void FillMatrix(float mat[], int m1, int m2){
    for (int i=0; i<m1; i++){
        for (int j=0; j<m2; j++){
            mat[i*m1+j] = static_cast<float>(rand())/static_cast<float>(RAND_MAX) * 10.0;
        }
    }
}
void PrintMatrix(float mat[], int m1, int m2){
    for (int i=0; i<m1; i++){
        for (int j=0; j<m2; j++){
            printf("%f,", mat[i*m1+j]);
        }
        printf("\n");
    }
}


int main(int argc, char *argv[]){
    int m1 = 10;
    int m2;
    int n1 = m2 = 10;
    int n2 = 10;
    float matrix_a[m1][m2];
    float matrix_b[n1][n2];
    FillMatrix((float*)matrix_a, m1, m2);
    FillMatrix((float*)matrix_b, n1, n2);
    float *res = MatrixMulCPU((float*)matrix_a, m1, m2, (float*)matrix_b, n1, n2);
    //float *res = MatrixMultGPU0((float*)matrix_a, m1, m2, (float*)matrix_b, n1, n2);
    PrintMatrix(res, m1, m2);
    delete res;
}
